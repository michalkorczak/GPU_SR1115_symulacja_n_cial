#include "hip/hip_runtime.h"
#include "../include/BarnesHut.cuh"

// calculate maximum of doubles atomically
__device__ void atomicMaxDouble(double* address, double val) {
  unsigned long long* address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long old = *address_as_ull, assumed;
  unsigned long long new_val = *reinterpret_cast<unsigned long long*>(&val);

  while (old < new_val) {
    old = atomicCAS(address_as_ull, assumed, new_val);
  }
}

__device__ void atomicMinDouble(double* address, double val) {
  unsigned long long* address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long old = *address_as_ull, assumed;
  unsigned long long new_val = *reinterpret_cast<unsigned long long*>(&val);

  while (old > new_val) {
    old = atomicCAS(address_as_ull, assumed, new_val);
  }
}

__global__ void computeBoundingBox(const Bodies& bodies, int n, double* bounds) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) {
    return;
  }

  extern __shared__ double shared_data[];

  double* x_min = shared_data;
  double* x_max = shared_data + blockDim.x;
  double* y_min = shared_data + 2 * blockDim.x;
  double* y_max = shared_data + 3 * blockDim.x;
  double* z_min = shared_data + 4 * blockDim.x;
  double* z_max = shared_data + 5 * blockDim.x;

  if (idx < n) {
    x_min[threadIdx.x] = bodies.position[idx].x;
    x_max[threadIdx.x] = bodies.position[idx].x;
    y_min[threadIdx.x] = bodies.position[idx].y;
    y_max[threadIdx.x] = bodies.position[idx].y;
    z_min[threadIdx.x] = bodies.position[idx].z;
    z_max[threadIdx.x] = bodies.position[idx].z;
  } else {
    // set to extreme values to not interfere with the reduction
    x_min[threadIdx.x] = DBL_MAX;
    x_max[threadIdx.x] = -DBL_MAX;
    y_min[threadIdx.x] = DBL_MAX;
    y_max[threadIdx.x] = -DBL_MAX;
    z_min[threadIdx.x] = DBL_MAX;
    z_max[threadIdx.x] = -DBL_MAX;
  }

  __syncthreads();

  // reduction
  // basically a tournament tree for comparing the values
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      x_min[threadIdx.x] = fmin(x_min[threadIdx.x], x_min[threadIdx.x + stride]);
      x_max[threadIdx.x] = fmax(x_max[threadIdx.x], x_max[threadIdx.x + stride]);
      y_min[threadIdx.x] = fmin(y_min[threadIdx.x], y_min[threadIdx.x + stride]);
      y_max[threadIdx.x] = fmax(y_max[threadIdx.x], y_max[threadIdx.x + stride]);
      z_min[threadIdx.x] = fmin(z_min[threadIdx.x], z_min[threadIdx.x + stride]);
      z_max[threadIdx.x] = fmax(z_max[threadIdx.x], z_max[threadIdx.x + stride]);
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    atomicMinDouble(&bounds[0], x_min[0]);
    atomicMaxDouble(&bounds[1], x_max[0]);
    atomicMinDouble(&bounds[2], y_min[0]);
    atomicMaxDouble(&bounds[3], y_max[0]);
  }
}

__device__ void insertBody(TreeNode* node, const Bodies& bodies, int bodyIdx, double x_min, double x_max, double y_min,
                           double y_max, double z_min, double z_max) {
  double x = bodies.position[bodyIdx].x;
  double y = bodies.position[bodyIdx].y;
  double z = bodies.position[bodyIdx].z;

  if (node->isLeaf) {
    if (node->bodyIndex == -1) {
      node->bodyIndex = bodyIdx;
    } else {
      double x_mid = (node->x_min + node->x_max) / 2;
      double y_mid = (node->y_min + node->y_max) / 2;
      double z_mid = (node->z_min + node->z_max) / 2;

      bool x_lower = x < x_mid;
      bool y_lower = y < y_mid;
      bool z_lower = z < z_mid;

      int childIndex = (x_lower ? 0 : 1) + (y_lower ? 0 : 2) + (z_lower ? 0 : 4);

      insertBody(node->children[childIndex], bodies, bodyIdx, x_min, x_max, y_min, y_max, z_min, z_max);
    }
  }
}

__global__ void buildTree(TreeNode* nodes, Bodies& bodies, int n, double x_min, double x_max, double y_min,
                          double y_max, double z_min, double z_max) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) {
    return;
  }

  TreeNode* node = &nodes[0];

  insertBody(node, bodies, idx, x_min, x_max, y_min, y_max, z_min, z_max);
}

__global__ void computeCentersOfMass(TreeNode* nodes, const Bodies& bodies, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) {
    return;
  }

  TreeNode* node = &nodes[idx];

  if (node->isLeaf) {
    // for leaf nodes, the center of mass is the body itself
    int bodyIdx = node->bodyIndex;
    node->cx = bodies.position[bodyIdx].x;
    node->cy = bodies.position[bodyIdx].y;
    node->cz = bodies.position[bodyIdx].z;
    node->mass = bodies.mass[bodyIdx];
  } else {
    // for internal nodes, the center of mass is the weighted average of the children's center of mass
    double totalMass = 0.0;
    double cx = 0.0, cy = 0.0, cz = 0.0;

    // calculate center of mass
    for (int i = 0; i < 8; i++) {
      TreeNode* child = node->children[i];
      if (node->children[i] != nullptr) {
        totalMass += node->children[i]->mass;
        cx += child->cx * child->mass;
        cy += child->cy * child->mass;
        cz += child->cz * child->mass;
      }
    }

    if (totalMass > 0.0) {
      node->cx = cx / totalMass;
      node->cy = cy / totalMass;
      node->cz = cz / totalMass;
    }
    node->mass = totalMass;
  }
}

__device__ void computeForcesRecursively(TreeNode* node, const Bodies& bodies, int idx, double mass, double x, double y,
                                         double z, double* fx, double* fy, double* fz, double theta) {
  if (node == nullptr) {
    return;
  }

  // calculate distance between the body and the center of mass of the node
  double dx = node->cx - x;
  double dy = node->cy - y;
  double dz = node->cz - z;
  double distance = sqrt(dx * dx + dy * dy + dz * dz);

  if (node->isLeaf) {
    if (node->bodyIndex != idx) {
      // calculate force between the body and the node
      double distanceSquared = distance * distance;
      double distanceCubed = distanceSquared * distance;
      double acceleration = G * node->mass / distanceCubed;

      *fx += dx * acceleration;
      *fy += dy * acceleration;
      *fz += dz * acceleration;
    }
  } else {
    if (distance / (node->x_max - node->x_min) < theta) {
      // if the ratio of the distance to the size of the node is less than theta, use the center of mass as a single
      // body
      double distanceSquared = distance * distance;
      double distanceCubed = distanceSquared * distance;
      double acceleration = G * node->mass / distanceCubed;

      *fx += dx * acceleration;
      *fy += dy * acceleration;
      *fz += dz * acceleration;
    } else {
      // otherwise, recursively calculate the forces from the children
      for (int i = 0; i < 8; i++) {
        computeForcesRecursively(node->children[i], bodies, idx, mass, x, y, z, fx, fy, fz, theta);
      }
    }
  }
}

__global__ void computeForces(TreeNode* nodes, const Bodies& bodies, double* forces_x, double* forces_y,
                              double* forces_z, int n, double theta) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) {
    return;
  }

  double fx = 0.0, fy = 0.0, fz = 0.0;
  double x = bodies.position[idx].x;
  double y = bodies.position[idx].y;
  double z = bodies.position[idx].z;
  double mass = bodies.mass[idx];

  // transverse the tree to calculate the forces
  TreeNode* node = &nodes[0];
  computeForcesRecursively(node, bodies, idx, mass, x, y, z, &fx, &fy, &fz, theta);
}

__global__ void updateBodies(Bodies bodies, int n, double* fx, double* fy, double* fz, double dt) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n) {
    return;
  }

  const double mass = bodies.mass[idx];

  bodies.acceleration[idx].x = fx[idx] / mass;
  bodies.acceleration[idx].y = fy[idx] / mass;
  bodies.acceleration[idx].z = fz[idx] / mass;

  bodies.velocity[idx].x += bodies.acceleration[idx].x * dt;
  bodies.velocity[idx].y += bodies.acceleration[idx].y * dt;
  bodies.velocity[idx].z += bodies.acceleration[idx].z * dt;

  bodies.position[idx].x += bodies.velocity[idx].x * dt;
  bodies.position[idx].y += bodies.velocity[idx].y * dt;
  bodies.position[idx].z += bodies.velocity[idx].z * dt;
}
