#include "../include/initialization.cuh"
#include <fstream>
#include <nlohmann/json.hpp>

void initializeBodies(Bodies bodies, int numberOfBodies, double spreadX,
                        double spreadY, double spreadZ) {
    double mass = 10e15;

  for (int i = 0; i < numberOfBodies; i++) {
    // initialize the bodies randomly spread around 0,0
    double x = static_cast<double>(rand()) / static_cast<double>(RAND_MAX) - 0.5;
    double y = static_cast<double>(rand()) / static_cast<double>(RAND_MAX) - 0.5;
    double z = static_cast<double>(rand()) / static_cast<double>(RAND_MAX) - 0.5;
    bodies.position[i].x = x * spreadX;
    bodies.position[i].y = x * spreadY;
    bodies.position[i].z = x * spreadZ;

    bodies.velocity[i].x = 0;
    bodies.velocity[i].y = 0;
    bodies.velocity[i].z = 0;

    bodies.mass[i] = mass;

    bodies.acceleration[i].x = 0;
    bodies.acceleration[i].y = 0;
    bodies.acceleration[i].z = 0;
  }
}


using json = nlohmann::json;
Config parseConfig(const int argc, const char** argv) {
  Config config;

  if (argc <= 1) {
    return config;
  }

  if (argc > 1) {
    if (strcmp(argv[1], "--help") == 0 || strcmp(argv[1], "-h") == 0) {
      std::cout << "Usage: " << argv[0] << " [number of bodies] [iterations] [save interval] [dt] [output filename]" << std::endl;
      exit(0);
    } else if (strcmp(argv[1], "--config") == 0 || strcmp(argv[1], "-c") == 0) {
      std::ifstream file(argv[2]);
      if (file.is_open()) {
        json configJson;
        file >> configJson;
        file.close();

        config.numberOfBodies = configJson["numberOfBodies"];
        config.iterations = configJson["iterations"];
        config.saveInterval = configJson["saveInterval"];
        config.dt = configJson["dt"];
        config.outputFilename = configJson["outputFilename"];
      } else {
        std::cout << "Error opening file." << std::endl;
        exit(0);
      }
    } else {
      config.numberOfBodies = atoi(argv[1]);
    }

    if (argc > 1) {
      config.numberOfBodies = atoi(argv[1]);
    }

    if (argc > 2) {
      config.iterations = atoi(argv[2]);
    }

    if (argc > 3) {
      config.saveInterval = atoi(argv[3]);
    }

    if (argc > 4) {
      config.dt = atof(argv[4]);
    }

    if (argc > 5) {
      config.outputFilename = argv[5];
    }
  }

  return config;
}

int getCudaBlockSize() {
  int deviceCount;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  if (err != hipSuccess) {
    std::cerr << "Error getting device count: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }

  if (deviceCount == 0) {
    std::cerr << "No CUDA devices found" << std::endl;
    exit(1);
  } else {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Found CUDA device: " << prop.name << std::endl;
    std::cout << "Block size: " << prop.maxThreadsPerBlock << std::endl;
    return prop.maxThreadsPerBlock;
  }
}