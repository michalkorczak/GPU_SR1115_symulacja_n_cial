#include "hip/hip_runtime.h"
#include "../include/BarnesHut.cuh"
#include "../include/file_operations.cuh"
#include "../include/initialization.cuh"
#include <iostream>

// #define BLOCK_SIZE 256
#define G 6.67430e-11 // Gravitational constant

void cleanup(Bodies& bodies, TreeNode* nodes) {
  hipFree(bodies.mass);
  hipFree(bodies.position);
  hipFree(bodies.velocity);
  hipFree(bodies.acceleration);

  hipFree(nodes);
}

int main(const int argc, const char** argv) {
  srand(time(NULL));
  Config config = parseConfig(argc, argv);

  int blockSize = getCudaBlockSize();

  // GPU bodies
  double* gpu_buffer;
  hipMalloc(&gpu_buffer, config.numberOfBodies * (sizeof(double) * 10));
  Bodies gpu_bodies = {(double*)gpu_buffer, (double3*)(gpu_buffer + config.numberOfBodies),
                       (double3*)(gpu_buffer + 3 * config.numberOfBodies), (double3*)(gpu_buffer + 5 * config.numberOfBodies)};

  // CPU bodies
  double* cpu_buffer = (double*)malloc(config.numberOfBodies * sizeof(double) * 10);
  Bodies cpu_bodies = {(double*)cpu_buffer, (double3*)(cpu_buffer + config.numberOfBodies),
                       (double3*)(cpu_buffer + 4 * config.numberOfBodies),
                       (double3*)(cpu_buffer + 7 * config.numberOfBodies)};

  initializeBodies(cpu_bodies, config.numberOfBodies);


  hipMemcpy(gpu_buffer, cpu_buffer, config.numberOfBodies * (sizeof(double) * 10), hipMemcpyHostToDevice);

  int maxNodes = 8 * config.numberOfBodies;
  TreeNode* nodes;
  hipMalloc((void**)&nodes, sizeof(TreeNode) * maxNodes);

  TreeNode* nodesCPU = (TreeNode*)malloc(config.numberOfBodies * sizeof(TreeNode) * maxNodes);

  double cpuBounds[6] = {0};

  double bounds[6];
  hipMalloc((void**)bounds, sizeof(double));

  hipMemcpy(bounds, cpuBounds, 6 * sizeof(double), hipMemcpyHostToDevice);

  int numberOfBlocks = (config.numberOfBodies + blockSize - 1) / blockSize;

  double *fx, *fy, *fz;
  hipMalloc((void**)&fx, sizeof(double) * config.numberOfBodies);
  hipMalloc((void**)&fy, sizeof(double) * config.numberOfBodies);
  hipMalloc((void**)&fz, sizeof(double) * config.numberOfBodies);

  for (int i = 0; i < config.iterations; i++) {
    computeBoundingBox<<<numberOfBlocks, blockSize>>>(gpu_bodies, config.numberOfBodies, bounds);
    hipDeviceSynchronize();
    buildTree<<<numberOfBlocks, blockSize>>>(nodes, gpu_bodies, config.numberOfBodies, bounds[0], bounds[1], bounds[2],
                                             bounds[3], bounds[4], bounds[5]);
    hipDeviceSynchronize();
    computeForces<<<numberOfBlocks, blockSize>>>(nodes, gpu_bodies, fx, fy, fz, config.numberOfBodies, 10);
    hipDeviceSynchronize();
    updateBodies<<<numberOfBlocks, blockSize>>>(gpu_bodies, config.numberOfBodies, fx, fy, fz, config.dt);

    if (config.saveInterval > 0 && i % config.saveInterval == 0) {
      hipDeviceSynchronize();
      hipMemcpy(cpu_bodies.mass, gpu_bodies.mass, config.numberOfBodies * sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(cpu_bodies.acceleration, gpu_bodies.acceleration, config.numberOfBodies * (sizeof(double) * 3),
                 hipMemcpyDeviceToHost);
      hipMemcpy(cpu_bodies.position, gpu_bodies.position, config.numberOfBodies * (sizeof(double) * 3),
                 hipMemcpyDeviceToHost);
      hipMemcpy(cpu_bodies.velocity, gpu_bodies.velocity, config.numberOfBodies * (sizeof(double) * 3),
                 hipMemcpyDeviceToHost);
      writeFile(config.outputFilename, &cpu_bodies, i * config.dt, config.numberOfBodies, i > config.saveInterval);
    }
  }

  cleanup(gpu_bodies, nodes);
  hipFree(fx);
  hipFree(fy);
  hipFree(fz);
  hipFree(bounds);
  free(cpu_buffer);
}