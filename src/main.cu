#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/structures.cuh"
#include "../include/initialization.cuh"

#define BLOCK_SIZE 256
#define G 6.67430e-11            // Gravitational constant

void initializeBodies(Bodies& bodies, int numberOfBodies) {
  initializeRandomly(bodies, numberOfBodies);
}

__global__ void computeBodyAcceleration(float2* position, float2* accelerations, float* masses, int n) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  if (index >= n) {
    return;
  }

  const auto this_position = position[index];

  accelerations[index].x = 0;
  accelerations[index].y = 0;

  // this is just a prototype without any software optimization - 2 pair method
  for (int i = 0; i < n; i++) {
    if (i != index) {
      float2 distance;
      distance.x = position[i].x - this_position.x;
      distance.y = position[i].y - this_position.y;

      float distanceSquared = distance.x * distance.x + distance.y * distance.y + 0.01;
      float distanceCubed = distanceSquared * sqrt(distanceSquared);

      float acceleration = G * masses[i] / distanceCubed;

      accelerations[index].x += distance.x * acceleration;
      accelerations[index].y += distance.y * acceleration;
    }
  }
}

__global__ void computeNewPositionAndSpeed(float2* positions, float2* velocities, float2* accelerations, float* masses,
                                           float dt, int n) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  if (index >= n) {
    return;
  }

  velocities[index].x += accelerations[index].x * dt;
  velocities[index].y += accelerations[index].y * dt;

  positions[index].x += velocities[index].x * dt;
  positions[index].y += velocities[index].y * dt;
}

int main(const int argc, const char** argv) {
  srand(time(NULL));

  int numberOfBodies = 1000; // default value
  if (argc > 1) {
    numberOfBodies = atoi(argv[1]);
  }

  float iterations = 1000;
  if (argc > 2) {
    iterations = atoi(argv[2]);
  }

  int printInterval = 100;
  if (argc > 3) {
    printInterval = atoi(argv[3]);
  }

  float dt = 0.01f;
  if (argc > 4) {
    dt = atof(argv[4]);
  }

  int numberOfBlocks = (numberOfBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;

  float* cpu_buffer = (float*)malloc(
      numberOfBodies * (sizeof(float) * 7)); // 7 floats per body - 1 mass + 2 position + 2 velocity + 2 acceleration
  Bodies cpu_bodies = {(float*)cpu_buffer, (float2*)(cpu_buffer + numberOfBodies),
                   (float2*)(cpu_buffer + 3 * numberOfBodies), (float2*)(cpu_buffer + 5 * numberOfBodies)};

  initializeBodies(cpu_bodies, numberOfBodies);

  float* gpu_buffer;
  hipMalloc(&gpu_buffer, numberOfBodies * (sizeof(float) * 7));
  Bodies gpu_bodies = {(float*)gpu_buffer, (float2*)(gpu_buffer + numberOfBodies),
                   (float2*)(gpu_buffer + 3 * numberOfBodies), (float2*)(gpu_buffer + 5 * numberOfBodies)};

  for (int i = 0; i < iterations; i++) {
    hipMemcpy(gpu_buffer, cpu_buffer, numberOfBodies * (sizeof(float) * 7), hipMemcpyHostToDevice);

    computeBodyAcceleration<<<numberOfBlocks, BLOCK_SIZE>>>(gpu_bodies.position, gpu_bodies.acceleration, gpu_bodies.mass,
                                                            numberOfBodies);
    hipDeviceSynchronize();

    computeNewPositionAndSpeed<<<numberOfBlocks, BLOCK_SIZE>>>(gpu_bodies.position, gpu_bodies.velocity, gpu_bodies.acceleration,
                                                               gpu_bodies.mass, dt, numberOfBodies);
    hipDeviceSynchronize();
    hipMemcpy(cpu_buffer, gpu_buffer, numberOfBodies * (sizeof(float) * 7), hipMemcpyDeviceToHost);


    if (printInterval > 0 && i % printInterval == 0)
    {
      std::cout << "ITERATION: " << i + 1 << "----------------" << std::endl;
      for (int j = 0; j < numberOfBodies; j++) {
        std::cout << "Number: " << j << " - x: " << cpu_bodies.position[j].x << " - y: " << cpu_bodies.position[j].y << std::endl;
      }
    }
  }

  hipFree(gpu_buffer);
}